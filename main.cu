
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    int N = 10;
    int a[N], b[N], c[N];

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<1,N>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}